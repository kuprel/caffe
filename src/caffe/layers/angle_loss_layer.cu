#include "hip/hip_runtime.h"
#include <vector>
#include <math.h>

#include "caffe/layer.hpp"
#include "caffe/util/io.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/vision_layers.hpp"

namespace caffe {

template <typename Dtype>
void AngleLossLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    vector<Blob<Dtype>*>* top) {
  // int n = bottom[0]->num();
  // int d = bottom[0]->count()/n;
  // const Dtype *u, *v;
  // Dtype *t = (*top)[0]->mutable_gpu_data();
  // Dtype L = 0;
  // for (int i=0; i<n; ++i) {
  //   u = bottom[0]->gpu_data() + i*d;
  //   v = bottom[1]->gpu_data() + i*d;
  //   caffe_gpu_dot(d, u, v, &L);
  //   L = acos(L);
  // }
  // *t = L/n;
  Forward_cpu(bottom, top);
}

template <typename Dtype>
void AngleLossLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, vector<Blob<Dtype>*>* bottom) {
  // for (int i = 0; i < 2; ++i) {
  //   if (propagate_down[i]) {
  //     caffe_gpu_scale(
  //       (*bottom)[i]->count(),
  //       top[0]->gpu_diff()[0] / (*bottom)[i]->num(),
  //       (*bottom)[1-i]->gpu_data(),
  //       (*bottom)[i]->mutable_gpu_diff()); 
  //   }
  // }
  Backward_cpu(top, propagate_down, bottom);
}

INSTANTIATE_CLASS(AngleLossLayer);

}  // namespace caffe
