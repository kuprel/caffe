#include "hip/hip_runtime.h"
#include <algorithm>
#include <cfloat>
#include <vector>

#include "thrust/device_vector.h"

#include "caffe/layer.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/vision_layers.hpp"

namespace caffe {

template <typename Dtype>
__global__ void kernel_channel_subtract(const int num, const int channels,
    const int spatial_dim, Dtype* data, const Dtype* channel_max) {
  CUDA_KERNEL_LOOP(index, num * spatial_dim) {
    int n = index / spatial_dim;
    int s = index % spatial_dim;
    for (int c = 0; c < channels; ++c) {
      data[(n * channels + c) * spatial_dim + s] -= channel_max[index];
    }
  }
}

template <typename Dtype>
__global__ void kernel_channel_dot(const int num, const int channels,
    const int spatial_dim, const Dtype* data_1, const Dtype* data_2,
    Dtype* channel_dot) {
  CUDA_KERNEL_LOOP(index, num * spatial_dim) {
    int n = index / spatial_dim;
    int s = index % spatial_dim;
    Dtype dot = 0;
    for (int c = 0; c < channels; ++c) {
      dot += (data_1[(n * channels + c) * spatial_dim + s]
          * data_2[(n * channels + c) * spatial_dim + s]);
    }
    channel_dot[index] = dot;
  }
}

template <typename Dtype>
void NormalizeLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    vector<Blob<Dtype>*>* top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = (*top)[0]->mutable_gpu_data();
  Dtype* squared_data = squared_.mutable_gpu_data();
  Dtype normsqr;
  int n = bottom[0]->num();
  int d = bottom[0]->count() / n;
  caffe_gpu_powx(n*d, bottom_data, Dtype(2), squared_data);
  for (int i=0; i<n; ++i) {
    caffe_gpu_asum<Dtype>(d, squared_data+i*d, &normsqr);
    caffe_gpu_scale<Dtype>(d, pow(normsqr, -0.5), bottom_data+i*d, top_data+i*d);
  }
}

template <typename Dtype>
void NormalizeLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, vector<Blob<Dtype>*>* bottom) {
  const Dtype* top_diff = top[0]->gpu_diff();
  const Dtype* top_data = top[0]->gpu_data();
  const Dtype* bottom_data = (*bottom)[0]->gpu_data();
  Dtype* bottom_diff = (*bottom)[0]->mutable_gpu_diff();
  int n = top[0]->num();
  int d = top[0]->count() / n;
  Dtype a;
  for (int i=0; i<n; ++i) {
    caffe_gpu_dot(d, top_data+i*d, top_diff+i*d, &a);
    caffe_gpu_scale(d, a, top_data+i*d, bottom_diff+i*d);
    caffe_gpu_sub(d, top_diff+i*d, bottom_diff+i*d, bottom_diff+i*d);
    caffe_gpu_dot(d, bottom_data+i*d, bottom_data+i*d, &a);
    caffe_gpu_scale(d, Dtype(pow(a, -0.5)), bottom_diff+i*d, bottom_diff+i*d);
  }
}

INSTANTIATE_CLASS(NormalizeLayer);


}  // namespace caffe
